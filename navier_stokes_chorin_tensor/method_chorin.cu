#include <hip/hip_runtime.h>
#include <hipblas.h>

__half* dev_u, * dev_v, * dev_u_prev, * dev_v_prev, * dev_p, * dev_p_prev, * dev_D_x, * dev_D_y, * dev_D_xx, * dev_D_yy;

__constant__ float dev_dx, dev_dy, dev_dt, dev_mu;

void allocate_2d_array_on_GPU(__half* u, __half* v, __half* u_prev, __half* v_prev, __half* p, __half* p_prev, __half*& D_x, __half*& D_y, __half*& D_xx, __half*& D_yy, int Nx, int Ny, __half dx, __half dy, __half dt, __half mu)
{
	//��������� ������ �� device
	hipMalloc((void**)&dev_u, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_v, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_u_prev, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_v_prev, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_p, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_p_prev, Nx * Ny * sizeof(__half));

	hipMalloc((void**)&dev_D_x, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_D_y, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_D_xx, Nx * Ny * sizeof(__half));
	hipMalloc((void**)&dev_D_yy, Nx * Ny * sizeof(__half));

	//����������� �������� �� ��� � ������ device
	hipMemcpy(dev_u, &u[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_v, &v[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_u_prev, &u_prev[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_v_prev, &v_prev[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, &p[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_p_prev, &p_prev[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);

	hipMemcpy(dev_D_x, &D_x[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_D_y, &D_y[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_D_xx, &D_xx[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
	hipMemcpy(dev_D_yy, &D_yy[0], Nx * Ny * sizeof(__half), hipMemcpyHostToDevice);
}

void free_2d_array_on_GPU()
{
    hipFree(dev_u);
    hipFree(dev_v);
    hipFree(dev_u_prev);
    hipFree(dev_v_prev);
    hipFree(dev_p);
    hipFree(dev_p_prev);
    hipFree(dev_D_x);
    hipFree(dev_D_y);
    hipFree(dev_D_xx);
    hipFree(dev_D_yy);
}

void calc_advect(__half* u, __half* v, __half* u_prev, __half* v_prev, __half*& D_x, __half*& D_y, int Nx, int Ny)
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float alpha = 1.0f, beta = 0.0f;

	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nx, Ny, Nx, &alpha, dev_D_x, HIP_R_16F, Nx, dev_u_prev, HIP_R_16F, Nx, &beta, dev_u, HIP_R_16F, Nx, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);

}


void copy_GPU_to_host(__half* u, __half* v, int Nx, int Ny)
{
	hipMemcpy(u, dev_u, Nx * Ny * sizeof(__half), hipMemcpyDeviceToHost);
	hipMemcpy(v, dev_v, Nx * Ny * sizeof(__half), hipMemcpyDeviceToHost);
}